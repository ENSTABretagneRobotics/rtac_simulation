#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;

#include <rtac_base/types/VectorView.h>
#include <rtac_base/types/Image.h>
using namespace rtac::types;

#include <rtac_base/cuda/texture_utils.h>

#include <rtac_display/Display.h>
#include <rtac_display/GLVector.h>
#include <rtac_display/renderers/ImageRenderer.h>
using namespace rtac::display;

#include <rtac_simulation/common.h>
#include <rtac_simulation/helpers/receiver_factories.h>
#include <rtac_simulation/PolarKernel2D.h>
using namespace rtac::simulation;

__global__ void render_kernel(rtac::types::ImageView<float> out, 
                              PolarKernelView2D<float> kernel,
                              float bearingSpan, float rangeSpan)
{
    float range = rangeSpan * (((float)blockIdx.x) / (out.height() - 1) - 0.5f);
    for(auto w = threadIdx.x; w < out.width(); w += blockDim.x) {
        float bearing = bearingSpan * (((float)w) / (out.width() - 1) - 0.5f);

        float value = kernel(range, bearing);
        //value = 0.5f * (value + 1.0f);
        value = value * value;

        out(blockIdx.x, w) = value;
    }
}

int main()
{
    float bearingResolution = 0.6f;
    float pulseLength       = 0.04f;
    float wavelength        = 1500.0 / 1.2e6;

    auto kernel = simple_polar_kernel<float>(bearingResolution,
                                             pulseLength, wavelength);
    rtac::display::Shape shape({kernel->texture().width(),
                                kernel->texture().height()});
    cout << "Kernel shape : " << shape << endl;

    Display display;
    float fps = 60.0;
    display.limit_frame_rate(fps);
    display.enable_frame_counter();
    
    GLVector<float> data(shape.area());
    {
        auto ptr = data.map_cuda();
        render_kernel<<<shape.height, 512>>>(
            rtac::types::ImageView<float>(shape,
                rtac::types::VectorView<float>(data.size(), ptr)),
            kernel->view(),
            kernel->bearing_span(), 
            kernel->range_span());
        hipDeviceSynchronize();
        CUDA_CHECK_LAST();
    }
    
    auto renderer = display.create_renderer<ImageRenderer>(View::New());
    renderer->enable_colormap();
    renderer->texture()->set_image(shape, data);

    while(!display.should_close()) {
        display.draw();
    }
    return 0;
}




