#include "hip/hip_runtime.h"
#include <rtac_simulation/SensorInstance1D.h>

#include <rtac_base/containers/Image.h>
#include <rtac_base/cuda/reductions.hcu>

namespace rtac { namespace simulation {

SensorInstance1D::SensorInstance1D(const SensorInfo::ConstPtr& info,
                                   const Pose& pose,
                                   float soundCelerity) :
    SensorInstance(info, pose, soundCelerity),
    info_(info)
{}

void SensorInstance1D::generate_psf_data()
{
    const auto& waveform = *waveform_;

    HostVector<Complex<float>> data(waveform.size());
    for(unsigned int i = 0; i < data.size(); i++) {
            data[i] = waveform[i];
    }

    waveformData_.set_image(data.size(), 1, (const float2*)data.data());
}

KernelView1D<Complex<float>> SensorInstance1D::kernel() const
{
    KernelView1D<Complex<float>> kernel;
    kernel.scaling_  = float2{1.0f / waveform_->duration()*soundCelerity_, 0.5f};
    kernel.function_ = waveformData_.texture();
    return kernel;
}

template <typename Tin, unsigned int BlockSize = 512>
__global__ void do_sparse_convolve_1d_f(VectorView<Complex<float>> out,
                                        Linspace<float> outRanges,
                                        const rtac::VectorView<const Tin>* bins,
                                        KernelView1D<Complex<float>> kernel)
{
    // shared memory does not play well with templates
    extern __shared__ __align__(sizeof(float)) unsigned char sharedMemory[];
    float* sdata = reinterpret_cast<float*>(sharedMemory);

    Complex<float> acc{0.0f,0.0f};
    for(auto i = threadIdx.x; i < bins[blockIdx.x].size(); i += blockDim.x)
    {
        auto datum = bins[blockIdx.x][i];
        // This multiplies the datum with the point spread function evaluated
        // at the difference between the datum location and the output bin
        // location.
        acc += datum.value()*kernel(
            datum.travel()  - outRanges[blockIdx.x]);
    }
    
    //reduction of cuda::Complex must be performed independently on real and
    //imag parts for now (issues with volatile conversion TODO:fix this)
    sdata[threadIdx.x] = acc.real();
    __syncthreads();
    cuda::device::reduce_shared<float,BlockSize,rtac::cuda::Addition>(sdata, threadIdx.x);
    if(threadIdx.x == 0)
        acc.real(sdata[0]);

    sdata[threadIdx.x] = acc.imag();
    __syncthreads();
    cuda::device::reduce_shared<float,BlockSize,rtac::cuda::Addition>(sdata, threadIdx.x);

    if(threadIdx.x == 0) {
        acc.imag(sdata[0]);
        out[blockIdx.x] = acc;
    }
}

void SensorInstance1D::do_reduce(cuda::CudaVector<Complex<float>>& out,
    const cuda::CudaVector<VectorView<const SimSample1D>>& bins) const
{
    out.resize(this->size());

    static constexpr unsigned int BlockSize = 512;

    do_sparse_convolve_1d_f<<<out.size(), BlockSize, sizeof(float)*BlockSize>>>(
        out.view(),
        this->ranges(),
        bins.data(),
        this->kernel());

    hipDeviceSynchronize();
    CUDA_CHECK_LAST();
}


} //namespace simulation
} //namespace rtac
