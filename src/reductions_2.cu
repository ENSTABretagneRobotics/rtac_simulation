#include "hip/hip_runtime.h"
#include <rtac_simulation/reductions_2.h>

#include <rtac_base/types/Complex.h>
#include <rtac_base/cuda/reductions.hcu>
#include <rtac_simulation/SensorModel.h>
#include <rtac_simulation/SensorInstance.h>


namespace rtac { namespace simulation {

template <typename D, typename Tin, typename KT, unsigned int BlockSize = 512>
__global__ void do_sparse_convolve_2d_f(D out,
                                        const rtac::VectorView<const Tin>* bins,
                                        KernelView2D<KT> kernel)
{
    // shared memory does not play well with templates
    using T = float;
    extern __shared__ __align__(sizeof(T)) unsigned char sharedMemory[];
    T* sdata = reinterpret_cast<T*>(sharedMemory);

    Complex<T> acc{0.0f,0.0f};
    for(auto i = threadIdx.x; i < bins[blockIdx.y].size(); i += blockDim.x)
    {
        auto datum = bins[blockIdx.y][i];
        // This multiplies the datum with the point spread function evaluated
        // at the difference between the datum location and the output bin
        // location.
        acc += datum.value()*kernel(
            datum.bearing() - out.width_dim() [blockIdx.x],
            datum.travel()  - out.height_dim()[blockIdx.y]);
    }
    
    //reduction of cuda::Complex must be performed independently on real and
    //imag parts for now (issues with volatile conversion TODO:fix this)
    sdata[threadIdx.x] = acc.real();
    __syncthreads();
    cuda::device::reduce_shared<T,BlockSize,rtac::cuda::Addition>(sdata, threadIdx.x);
    if(threadIdx.x == 0)
        acc.real(sdata[0]);

    sdata[threadIdx.x] = acc.imag();
    __syncthreads();
    cuda::device::reduce_shared<T,BlockSize,rtac::cuda::Addition>(sdata, threadIdx.x);

    if(threadIdx.x == 0) {
        acc.imag(sdata[0]);
        out(blockIdx.y,blockIdx.x) = acc;
    }
}

void sparse_convolve_2d(SensorModel2D<Complex<float>, float>& out,
                        const cuda::DeviceVector<VectorView<const SimSample2D>>& bins)
{
    if(out.height() != bins.size()) {
        throw std::runtime_error("Inconsistent sizes");
    }

    static constexpr unsigned int BlockSize = 512;
    uint3 grid{out.width(), out.height(), 1};
    do_sparse_convolve_2d_f<<<grid, BlockSize, sizeof(float)*BlockSize>>>(
        out.data().view(), bins.data(), out.point_spread_function().view());
    hipDeviceSynchronize();
    CUDA_CHECK_LAST();
}

void sparse_convolve_2d(SensorModel2D_2<Complex<float>>& out,
                        const cuda::DeviceVector<VectorView<const SimSample2D>>& bins)
{
    if(out.height() != bins.size()) {
        throw std::runtime_error("Inconsistent sizes");
    }
    static constexpr unsigned int BlockSize = 512;
    uint3 grid{out.width(), out.height(), 1};

    if(out.point_spread_function()->is_complex()) {
        do_sparse_convolve_2d_f<<<grid, BlockSize, sizeof(float)*BlockSize>>>(
            out.data().view(), bins.data(),
            out.point_spread_function()->complex_cast()->kernel());
    }
    else {
        do_sparse_convolve_2d_f<<<grid, BlockSize, sizeof(float)*BlockSize>>>(
            out.data().view(), bins.data(),
            out.point_spread_function()->real_cast()->kernel());
    }

    hipDeviceSynchronize();
    CUDA_CHECK_LAST();
}

template <typename Tin, typename KT, unsigned int BlockSize = 512>
__global__ void do_sparse_convolve_2d_f(ImageView<Complex<float>> out,
                                        cuda::TextureVectorView<float> outBearings,
                                        Linspace<float> outRanges,
                                        const rtac::VectorView<const Tin>* bins,
                                        KernelView2D<KT> kernel)
{
    // shared memory does not play well with templates
    using T = float;
    extern __shared__ __align__(sizeof(T)) unsigned char sharedMemory[];
    T* sdata = reinterpret_cast<T*>(sharedMemory);

    Complex<T> acc{0.0f,0.0f};
    for(auto i = threadIdx.x; i < bins[blockIdx.y].size(); i += blockDim.x)
    {
        auto datum = bins[blockIdx.y][i];
        // This multiplies the datum with the point spread function evaluated
        // at the difference between the datum location and the output bin
        // location.
        acc += datum.value()*kernel(
            datum.bearing() - outBearings[blockIdx.x],
            datum.travel()  - outRanges[blockIdx.y]);
    }
    
    //reduction of cuda::Complex must be performed independently on real and
    //imag parts for now (issues with volatile conversion TODO:fix this)
    sdata[threadIdx.x] = acc.real();
    __syncthreads();
    cuda::device::reduce_shared<T,BlockSize,rtac::cuda::Addition>(sdata, threadIdx.x);
    if(threadIdx.x == 0)
        acc.real(sdata[0]);

    sdata[threadIdx.x] = acc.imag();
    __syncthreads();
    cuda::device::reduce_shared<T,BlockSize,rtac::cuda::Addition>(sdata, threadIdx.x);

    if(threadIdx.x == 0) {
        acc.imag(sdata[0]);
        out(blockIdx.y,blockIdx.x) = acc;
    }
}

void sparse_convolve_2d(SensorModel2D_Base& out,
                        const cuda::DeviceVector<VectorView<const SimSample2D>>& bins)
{
    if(out.height() != bins.size()) {
        throw std::runtime_error("Inconsistent sizes");
    }
    static constexpr unsigned int BlockSize = 512;
    uint3 grid{out.width(), out.height(), 1};

    auto& outTmp = dynamic_cast<SensorModel2D_Complex&>(out);

    //if(out.point_spread_function()->is_complex()) {
    //    do_sparse_convolve_2d_f<<<grid, BlockSize, sizeof(float)*BlockSize>>>(
    //        outTmp.data_view(), 
    //        out.bearings_view(),
    //        out.ranges(),
    //        bins.data(),
    //        outTmp.point_spread_function()->complex_cast()->kernel());
    //}
    //else {
    //    do_sparse_convolve_2d_f<<<grid, BlockSize, sizeof(float)*BlockSize>>>(
    //        outTmp.data_view(), 
    //        out.bearings_view(),
    //        out.ranges(),
    //        bins.data(),
    //        outTmp.point_spread_function()->real_cast()->kernel());
    //}
    do_sparse_convolve_2d_f<<<grid, BlockSize, sizeof(float)*BlockSize>>>(
        outTmp.data_view(), 
        out.bearings_view(),
        out.ranges(),
        bins.data(),
        outTmp.point_spread_function()->kernel());

    hipDeviceSynchronize();
    CUDA_CHECK_LAST();
}

void sparse_convolve_2d(Image<Complex<float>, cuda::DeviceVector>& out,
                        const SensorInstance& sensor,
                        const cuda::DeviceVector<VectorView<const SimSample2D>>& bins)
{
    if(sensor.height() != bins.size()) {
        throw std::runtime_error("Inconsistent sizes");
    }
    out.resize({sensor.width(), sensor.height()});

    static constexpr unsigned int BlockSize = 512;
    uint3 grid{out.width(), out.height(), 1};

    do_sparse_convolve_2d_f<<<grid, BlockSize, sizeof(float)*BlockSize>>>(
        out.view(),
        sensor.bearings_view(),
        sensor.ranges(),
        bins.data(),
        sensor.kernel());

    hipDeviceSynchronize();
    CUDA_CHECK_LAST();
}


} //namespace simulation
} //namespace rtac
