#include "hip/hip_runtime.h"
#include <rtac_simulation/Emitter.h>

namespace rtac { namespace simulation {

Emitter::Emitter(const cuda::DeviceVector<float3>& rayDirs,
                 Directivity::ConstPtr directivity,
                 const Pose& pose) :
    Antenna(directivity, pose),
    directions_(rayDirs),
    initialValues_(rayDirs.size())
{
    this->load_initial_values();
}

Emitter::Ptr Emitter::Create(const cuda::DeviceVector<float3>& rayDirs,
                             Directivity::ConstPtr directivity,
                             const Pose& pose)
{
    return Ptr(new Emitter(rayDirs, directivity, pose));
}

Emitter::Ptr Emitter::Create(float resolution,
                             float bearingAperture,
                             float elevationAperture,
                             Directivity::ConstPtr directivity,
                             const Pose& pose)
{
    return Create(generate_polar_directions(resolution,
                                            bearingAperture,
                                            elevationAperture),
                  directivity, pose); 
}

__global__ void load_emitter_initial_values(unsigned int size,
                                            Complex<float>* values,
                                            const float3* directions,
                                            DirectivityView directivity)
{
    auto tid = blockDim.x*blockIdx.x + threadIdx.x;
    if(tid < size) {
        values[tid] = directivity(directions[tid]);
    }
}

void Emitter::load_initial_values()
{
    load_emitter_initial_values<<<this->size() / 256 + 1, 256>>>(
        this->size(), initialValues_.data(), directions_.data(),
        this->directivity()->view());
    hipDeviceSynchronize();
    CUDA_CHECK_LAST();
}

cuda::DeviceVector<float3> Emitter::generate_polar_directions(float resolution,
                                                               float bearingAperture,
                                                               float elevationAperture)
{
    resolution        *= M_PI  / 180.0f;
    bearingAperture   *= M_PI  / 180.0f;
    elevationAperture *= M_PI  / 180.0f;

    //unsigned int idx = 0;
    unsigned int H   = (int)(elevationAperture / resolution) + 1;
    
    std::vector<float3> data;
    for(int h = 0; h < H; h++) {
        float phi = elevationAperture * (((float)h) / (H - 1) - 0.5f);
        int W = (int)(abs(std::cos(phi)) * bearingAperture   / resolution) + 1;
        for(int w = 0; w < W; w++) {
            float theta = bearingAperture * (((float)w) / (W - 1) - 0.5f);
            data.push_back(float3{std::cos(theta)*std::cos(phi),
                                  std::sin(theta)*std::cos(phi),
                                  std::sin(phi)});
        }
    }
    return cuda::DeviceVector<float3>(data);
}


} //namespace simulation
} //namespace rtac
